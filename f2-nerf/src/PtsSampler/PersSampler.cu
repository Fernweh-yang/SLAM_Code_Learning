#include "hip/hip_runtime.h"
//
// Created by ppwang on 2022/9/26.
//

#include "PersSampler.h"
#include "../Utils/Utils.h"
#define MAX_STACK_SIZE 48
#define MAX_OCT_INTERSECT_PER_RAY 1024
#define MAX_SAMPLE_PER_RAY 1024

#define OCC_WEIGHT_BASE 512
#define ABS_WEIGHT_THRES 0.01
#define REL_WEIGHT_THRES 0.1

#define OCC_ALPHA_BASE 32
#define ABS_ALPHA_THRES 0.02
#define REL_ALPHA_THRES 0.1

using Tensor = torch::Tensor;

inline __device__ void GetIntersection(const Wec3f& rays_o,
                                       const Wec3f& rays_d,
                                       const Wec3f& oct_center,
                                       float oct_side_len,
                                       float* near,
                                       float* far) {
  float tmp[3][2];
  float hf_len = oct_side_len * .5f;
#pragma unroll
  for (int i = 0; i < 3; i++) {
    if (rays_d[i] < 1e-6f && rays_d[i] > -1e-6f) {
      if (rays_o[i] > oct_center[i] - hf_len && rays_o[i] < oct_center[i] + hf_len) {
        tmp[i][0] = -1e6f; tmp[i][1] = 1e6f;
      }
      else {
        tmp[i][0] = 1e6f; tmp[i][1] = -1e6f;
      }
    }
    else if (rays_d[i] > 0) {
      tmp[i][0] = (oct_center[i] - hf_len - rays_o[i]) / rays_d[i];
      tmp[i][1] = (oct_center[i] + hf_len - rays_o[i]) / rays_d[i];
    }
    else {
      tmp[i][0] = (oct_center[i] + hf_len - rays_o[i]) / rays_d[i];
      tmp[i][1] = (oct_center[i] - hf_len - rays_o[i]) / rays_d[i];
    }
  }

  near[0] = fmaxf(near[0], fmaxf(tmp[0][0], fmaxf(tmp[1][0], tmp[2][0])));
  far[0] = fminf(far[0], fminf(tmp[0][1], fminf(tmp[1][1], tmp[2][1])));
}

template <bool FILL>
__global__ void FindRayOctreeIntersectionKernel(int n_rays, int max_oct_intersect_per_ray,
                                                uint8_t* search_order,
                                                Wec3f* rays_o_ptr, Wec3f* rays_d_ptr, Wec2f* bounds,
                                                int* oct_idx_counter, Wec2i* oct_idx_start_end_ptr,
                                                TreeNode* tree_nodes,
                                                int* oct_intersect_idx, Wec2f* oct_intersect_near_far,
                                                int* ) {
  int ray_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (ray_idx >= n_rays) {
    return;
  }
  // Add offsets
  const Wec3f& rays_o = rays_o_ptr[ray_idx];
  const Wec3f& rays_d = rays_d_ptr[ray_idx];
  Wec2i& oct_idx_start_end = oct_idx_start_end_ptr[ray_idx];
  // stack_info = stack_info + ray_idx * MAX_STACK_SIZE;
  int stack_info[MAX_STACK_SIZE];
  const float overall_near = bounds[ray_idx][0];
  const float overall_far  = bounds[ray_idx][1];

  int max_intersect_cnt = max_oct_intersect_per_ray;
  if (FILL) {
    max_intersect_cnt = oct_idx_start_end[1] - oct_idx_start_end[0];
    oct_intersect_idx = oct_intersect_idx + oct_idx_start_end[0];
    oct_intersect_near_far = oct_intersect_near_far + oct_idx_start_end[0];
  }

  int stack_ptr = 0;
  int intersect_cnt = 0;

  stack_info[0] = 0;  // Root octree node
  stack_info[1] = -1;

  int ray_st = (int(rays_d[0] > 0.f) << 2) | (int(rays_d[1] > 0.f) << 1) | (int(rays_d[2] > 0.f) << 0);
  search_order += ray_st * 8;
  while (stack_ptr >= 0 && intersect_cnt < max_intersect_cnt) {
    int u = stack_info[stack_ptr * 2]; // Octree node idx;
    const auto& node = tree_nodes[u];
    if (stack_info[stack_ptr * 2 + 1] == -1) {
      float cur_near = overall_near, cur_far = overall_far;
      GetIntersection(rays_o, rays_d, node.center, node.side_len, &cur_near, &cur_far);
      bool can_live_stack = cur_near < cur_far;

      if (can_live_stack) {
        int child_ptr = 0;
        while (child_ptr < 8 && node.childs[search_order[child_ptr]] < 0) {
          child_ptr++;
        }
        if (child_ptr < 8) {   // Has childs, push stack
          stack_info[stack_ptr * 2 + 1] = child_ptr;
          stack_ptr++;
          stack_info[stack_ptr * 2] = node.childs[search_order[child_ptr]];
          stack_info[stack_ptr * 2 + 1] = -1;
        }
        else {
          // Leaf node
          if (node.trans_idx >= 0) {
            if (FILL) {
              oct_intersect_idx[intersect_cnt] = u;
              oct_intersect_near_far[intersect_cnt][0] = cur_near;
              oct_intersect_near_far[intersect_cnt][1] = cur_far;
            }
            intersect_cnt++;
          }
          stack_ptr--;
        }
      }
      else {
        stack_ptr--;
      }
    }
    else {
      int child_ptr = stack_info[stack_ptr * 2 + 1] + 1;
      while (child_ptr < 8 && node.childs[search_order[child_ptr]] < 0) {
        child_ptr++;
      }
      if (child_ptr < 8) {
        stack_info[stack_ptr * 2 + 1] = child_ptr;
        stack_ptr++;
        stack_info[stack_ptr * 2] = node.childs[search_order[child_ptr]];
        stack_info[stack_ptr * 2 + 1] = -1;
      }
      else {
        stack_ptr--;
      }
    }
  }

  if (!FILL) {
    // Phase 1
    int idx_start = atomicAdd(oct_idx_counter, intersect_cnt);
    oct_idx_start_end[0] = idx_start;
    oct_idx_start_end[1] = idx_start + intersect_cnt;
  }
  else {
    // Phase 2
    oct_idx_start_end[1] = oct_idx_start_end[0] + intersect_cnt;
  }
}


void __device__ QueryFrameTransform(const TransInfo& trans,
                                    const Wec3f& cur_xyz,
                                    Wec3f* fill_xyz) {
  Wec4f cur_xyz_ext;
  cur_xyz_ext = cur_xyz.homogeneous();
  Eigen::Matrix<float, N_PROS, 1> transed_vals;
#pragma unroll
  for (int i = 0; i < N_PROS; i++) {
    Wec2f xz = trans.w2xz[i] * cur_xyz_ext;
    transed_vals(i, 0) = xz[0] / xz[1];
  }

  Wec3f weighted = trans.weight * transed_vals;
  *fill_xyz = weighted;
}

void __device__ QueryFrameTransformJac(const TransInfo& trans,
                                       const Wec3f& cur_xyz,
                                       Watrix33f* jac) {
  Wec4f cur_xyz_ext = cur_xyz.homogeneous();
  Eigen::Matrix<float, N_PROS, 3, Eigen::RowMajor> transed_jac;

#pragma unroll
  for (int i = 0; i < N_PROS; i++) {
    Wec2f xz = trans.w2xz[i] * cur_xyz_ext;
    Eigen::Matrix<float, 1, 2, Eigen::RowMajor> dv_dxz;
    dv_dxz(0, 0) = 1 / xz[1]; dv_dxz(0, 1) =-xz[0] / (xz[1] * xz[1]);
    transed_jac.block<1, 3>(i, 0) = dv_dxz * trans.w2xz[i].block<2, 3>(0, 0);
  }

  Watrix33f weighted_jac = trans.weight * transed_jac;
  *jac = weighted_jac;
}

template<bool FILL>
__global__ void RayMarchKernel(int n_rays, float sample_l, bool scale_by_dis,
                               Wec3f* rays_o_ptr, Wec3f* rays_d_ptr, float* rays_noise,
                               Wec2i* oct_idx_start_end_ptr, int* oct_intersect_idx, Wec2f* oct_intersect_near_far,
                               TreeNode* tree_nodes, TransInfo* transes,
                               Wec2i* pts_idx_start_end_ptr,
                               Wec3f* sampled_world_pts, Wec3f* sampled_pts, Wec3f* sampled_dirs, Wec3i* sampled_anchors,
                               float* sampled_dists, float* sampled_ts, int* sampled_oct_idx,
                               float* first_oct_dis) {
  int ray_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (ray_idx >= n_rays) {
    return;
  }

  rays_noise = rays_noise + ray_idx;
  const auto& rays_o = rays_o_ptr[ray_idx];
  const auto& rays_d = rays_d_ptr[ray_idx];
  const auto& oct_idx_start_end = oct_idx_start_end_ptr[ray_idx];
  oct_intersect_idx = oct_intersect_idx + oct_idx_start_end[0];
  oct_intersect_near_far = oct_intersect_near_far + oct_idx_start_end[0];
  auto& pts_idx_start_end = pts_idx_start_end_ptr[ray_idx];

  int pts_idx = 0;
  if (FILL) {
    int idx_end = pts_idx_start_end[0];
    int idx_cnt = pts_idx_start_end[1];
    pts_idx_start_end[0] = idx_end - idx_cnt;
    pts_idx_start_end[1] = idx_end;
    pts_idx = pts_idx_start_end[0];
    sampled_world_pts = sampled_world_pts + pts_idx;
    sampled_pts = sampled_pts + pts_idx;
    sampled_dirs = sampled_dirs + pts_idx;
    sampled_anchors = sampled_anchors + pts_idx;
    sampled_dists = sampled_dists + pts_idx;
    sampled_ts = sampled_ts + pts_idx;
    sampled_oct_idx = sampled_oct_idx + pts_idx;

    if (oct_idx_start_end[0] < oct_idx_start_end[1]) {
      first_oct_dis[ray_idx] = oct_intersect_near_far[0][0];
    }
    else {
      first_oct_dis[ray_idx] = 1e9f;
    }
  }
  int max_n_samples = FILL ? pts_idx_start_end[1] - pts_idx_start_end[0] : MAX_SAMPLE_PER_RAY;

  if (max_n_samples <= 0) {
    return;
  }

  int oct_ptr = 0;
  int pts_ptr = 0;
  int cur_oct_idx = oct_intersect_idx[0];
  float cur_march_step = 0.f;
  float exp_march_step = 0.f;

  int n_oct_nodes = oct_idx_start_end[1] - oct_idx_start_end[0];

  float cur_t = oct_intersect_near_far[0][0];
  float cur_far = oct_intersect_near_far[0][1];
  float cur_near = oct_intersect_near_far[0][0];
  Wec3f cur_xyz = rays_o + rays_d * cur_t;
  Wec3f nex_xyz;

  bool the_first_pts = true;
  while (pts_ptr < max_n_samples && oct_ptr < n_oct_nodes) {
    Wec3f fill_xyz = Wec3f::Zero();

    const auto& cur_node = tree_nodes[cur_oct_idx];

    // Get march step
    Watrix33f jac = Watrix33f::Zero();
    const auto& cur_trans = transes[cur_node.trans_idx];
    float cur_radius = (rays_o - cur_trans.center).norm() / cur_trans.dis_summary;
    float cur_radius_clip = fmaxf(cur_radius, 1.f);
    QueryFrameTransformJac(cur_trans, cur_xyz, &jac);
    Wec3f proj_xyz = jac * rays_d;
    float exp_march_step_warp = sample_l * rays_noise[pts_ptr];
    exp_march_step = exp_march_step_warp / (proj_xyz.norm() + 1e-6f);
    if (scale_by_dis) {
      exp_march_step *= cur_radius_clip;
    }

    cur_march_step = exp_march_step;

    // Do not consider the first point in sampling, because the first point has no randomness in training.
    if (FILL && !the_first_pts) {
      sampled_world_pts[pts_ptr] = cur_xyz;
      sampled_ts[pts_ptr] = cur_t;
      sampled_oct_idx[pts_ptr] = cur_oct_idx;
      sampled_dirs[pts_ptr] = rays_d;

      QueryFrameTransform(cur_trans, cur_xyz, &fill_xyz);
      sampled_dists[pts_ptr] = exp_march_step * (proj_xyz.norm() + 1e-6f);
      sampled_pts[pts_ptr] = fill_xyz;
      sampled_anchors[pts_ptr][0] = cur_node.trans_idx;
      sampled_anchors[pts_ptr][1] = cur_oct_idx;
    }
    if (!the_first_pts) {
      pts_ptr += 1;
    }

    while (cur_t + cur_march_step > cur_far) {
      oct_ptr++;
      if (oct_ptr >= n_oct_nodes) {
        break;
      }
      cur_oct_idx = oct_intersect_idx[oct_ptr];
      cur_near = oct_intersect_near_far[oct_ptr][0];
      cur_far = oct_intersect_near_far[oct_ptr][1];
      int ex_march_steps = ceilf(fmaxf((cur_near - cur_t) / exp_march_step, 1.f));
      cur_march_step = exp_march_step * float(ex_march_steps);
    }
    cur_t += cur_march_step;
    cur_xyz = rays_o + rays_d * cur_t;
    the_first_pts = false;
  }

  if (FILL) {
    pts_idx_start_end[1] = pts_idx_start_end[0] + pts_ptr;
  }
  else {
    pts_idx_start_end[0] = pts_ptr;
    pts_idx_start_end[1] = pts_ptr;
  }
}


SampleResultFlex PersSampler::GetSamples(const Tensor& rays_o_raw, const Tensor& rays_d_raw, const Tensor& bounds_raw) {
  Tensor rays_o = rays_o_raw.contiguous();
  Tensor rays_d = (rays_d_raw / torch::linalg_norm(rays_d_raw, 2, -1, true)).contiguous();

  int n_rays = rays_o.sizes()[0];
  Tensor bounds = torch::stack({ torch::full({n_rays}, global_near_, CUDAFloat),
                               torch::full({n_rays}, 1e8f, CUDAFloat) }, -1).contiguous();

  // First, find octree intersections
  Tensor oct_idx_counter = torch::zeros({1}, CUDAInt);
  Tensor oct_idx_start_end = torch::zeros({ n_rays, 2 }, CUDAInt);
  Tensor stack_info = torch::zeros({ n_rays * MAX_STACK_SIZE }, CUDAInt);

  CK_CONT(rays_o);
  CK_CONT(rays_d);
  CK_CONT(oct_idx_counter);
  CK_CONT(oct_idx_start_end);
  CK_CONT(stack_info);
  CK_CONT(bounds);
  CK_CONT(pers_octree_->tree_nodes_gpu_);
  CK_CONT(pers_octree_->pers_trans_gpu_);

  dim3 block_dim = LIN_BLOCK_DIM(n_rays);
  dim3 grid_dim = LIN_GRID_DIM(n_rays);

  FindRayOctreeIntersectionKernel<false><<<grid_dim, block_dim>>>(
      n_rays, max_oct_intersect_per_ray_,
      pers_octree_->node_search_order_.data_ptr<uint8_t>(),
      RE_INTER(Wec3f*, rays_o.data_ptr()),
      RE_INTER(Wec3f*, rays_d.data_ptr()),
      RE_INTER(Wec2f*, bounds.data_ptr()),
      oct_idx_counter.data_ptr<int>(), RE_INTER(Wec2i*, oct_idx_start_end.data_ptr()),
      RE_INTER(TreeNode*, pers_octree_->tree_nodes_gpu_.data_ptr()),
      nullptr, nullptr,
      stack_info.data_ptr<int>());

  int n_all_oct_intersect = oct_idx_counter.item<int>();
  Tensor oct_intersect_idx = torch::empty({ n_all_oct_intersect }, CUDAInt);
  Tensor oct_intersect_near_far = torch::empty({ n_all_oct_intersect, 2 }, CUDAFloat);

  FindRayOctreeIntersectionKernel<true><<<grid_dim, block_dim>>>(
      n_rays, max_oct_intersect_per_ray_,
      pers_octree_->node_search_order_.data_ptr<uint8_t>(),
      RE_INTER(Wec3f*, rays_o.data_ptr()),
      RE_INTER(Wec3f*, rays_d.data_ptr()),
      RE_INTER(Wec2f*, bounds.data_ptr()),
      oct_idx_counter.data_ptr<int>(), RE_INTER(Wec2i*, oct_idx_start_end.data_ptr()),
      RE_INTER(TreeNode*, pers_octree_->tree_nodes_gpu_.data_ptr()),
      oct_intersect_idx.data_ptr<int>(), RE_INTER(Wec2f*, oct_intersect_near_far.data_ptr()),
      stack_info.data_ptr<int>());


  // Second, do ray marching
  Tensor pts_idx_start_end = torch::zeros({ n_rays, 2 }, CUDAInt);

  Tensor rays_noise;
  if (global_data_pool_->mode_ == RunningMode::VALIDATE) {
    rays_noise = torch::ones({ MAX_SAMPLE_PER_RAY + n_rays + 10 }, CUDAFloat);
  }
  else {
    rays_noise = ((torch::rand({ MAX_SAMPLE_PER_RAY + n_rays + 10 }, CUDAFloat) - .5f) + 1.f).contiguous();
    float sampled_oct_per_ray = float(n_all_oct_intersect) / float(n_rays);
    global_data_pool_->sampled_oct_per_ray_ = global_data_pool_->sampled_oct_per_ray_ * .9f + sampled_oct_per_ray * .1f;
  }
  rays_noise.mul_(global_data_pool_->ray_march_fineness_);

  RayMarchKernel<false><<<grid_dim, block_dim>>>(
      n_rays, sample_l_, scale_by_dis_,
      RE_INTER(Wec3f*, rays_o.data_ptr()), RE_INTER(Wec3f*, rays_d.data_ptr()),
      rays_noise.data_ptr<float>(),
      RE_INTER(Wec2i*, oct_idx_start_end.data_ptr()), oct_intersect_idx.data_ptr<int>(), RE_INTER(Wec2f*, oct_intersect_near_far.data_ptr()),
      // unsigned char* occ_bits_tables,
      RE_INTER(TreeNode*, pers_octree_->tree_nodes_gpu_.data_ptr()),
      RE_INTER(TransInfo*, pers_octree_->pers_trans_gpu_.data_ptr()),
      RE_INTER(Wec2i*, pts_idx_start_end.data_ptr()),
      nullptr, nullptr, nullptr, nullptr, nullptr, nullptr, nullptr, nullptr
  );

  pts_idx_start_end.index_put_({Slc(), 0}, torch::cumsum(pts_idx_start_end.index({Slc(), 0}), 0));

  int n_all_pts = pts_idx_start_end.index({-1, 0}).item<int>();
  Tensor sampled_world_pts = torch::empty({ n_all_pts, 3 }, CUDAFloat);
  Tensor sampled_pts = torch::empty({ n_all_pts, 3 }, CUDAFloat);
  Tensor sampled_dirs = torch::empty({ n_all_pts, 3 }, CUDAFloat);
  Tensor sampled_anchors = torch::empty({ n_all_pts, 3 }, CUDAInt);
  Tensor sampled_dists = torch::empty({ n_all_pts }, CUDAFloat);
  Tensor sampled_t = torch::empty({ n_all_pts }, CUDAFloat);
  Tensor sampled_oct_idx = torch::full({ n_all_pts }, -1,CUDAInt).contiguous();
  Tensor first_oct_dis = torch::zeros({ n_rays, 1 }, CUDAFloat).contiguous();

  RayMarchKernel<true><<<grid_dim, block_dim>>>(
      n_rays, sample_l_, scale_by_dis_,
      RE_INTER(Wec3f*, rays_o.data_ptr()), RE_INTER(Wec3f*, rays_d.data_ptr()),
      rays_noise.data_ptr<float>(),
      RE_INTER(Wec2i*, oct_idx_start_end.data_ptr()), oct_intersect_idx.data_ptr<int>(), RE_INTER(Wec2f*, oct_intersect_near_far.data_ptr()),
      // unsigned char* occ_bits_tables,
      RE_INTER(TreeNode*, pers_octree_->tree_nodes_gpu_.data_ptr()),
      RE_INTER(TransInfo*, pers_octree_->pers_trans_gpu_.data_ptr()),
      RE_INTER(Wec2i*, pts_idx_start_end.data_ptr()),
      RE_INTER(Wec3f*, sampled_world_pts.data_ptr()),
      RE_INTER(Wec3f*, sampled_pts.data_ptr()),
      RE_INTER(Wec3f*, sampled_dirs.data_ptr()),
      RE_INTER(Wec3i*, sampled_anchors.data_ptr()),
      sampled_dists.data_ptr<float>(), sampled_t.data_ptr<float>(),
      sampled_oct_idx.data_ptr<int>(),
      first_oct_dis.data_ptr<float>()
  );

  return {
      sampled_pts,
      sampled_dirs,
      sampled_dists,
      sampled_t,
      sampled_anchors,
      pts_idx_start_end,
      first_oct_dis,
  };
}

__global__ void GetEdgeSamplesKernel(int n_pts, EdgePool* edge_pool, TransInfo* trans, int* edge_indices, Wec2f* edge_coords,
                                     Wec3f* out_pts, int* out_idx) {
  int pts_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (pts_idx >= n_pts) { return; }
  int edge_idx = edge_indices[pts_idx];
  edge_pool += edge_idx;
  Wec3f world_pts = edge_pool->center + edge_pool->dir_0 * edge_coords[pts_idx][0] + edge_pool->dir_1 * edge_coords[pts_idx][1];
  Wec3f warp_pts_a, warp_pts_b;
  int a = edge_pool->t_idx_a; int b = edge_pool->t_idx_b;
  QueryFrameTransform(trans[a], world_pts, &warp_pts_a);
  QueryFrameTransform(trans[b], world_pts, &warp_pts_b);

  out_pts[pts_idx * 2] = warp_pts_a;
  out_pts[pts_idx * 2 + 1] = warp_pts_b;
  out_idx[pts_idx * 2] = a;
  out_idx[pts_idx * 2 + 1] = b;
}

std::tuple<Tensor, Tensor> PersSampler::GetEdgeSamples(int n_pts) {
  int n_edges = pers_octree_->edge_pool_.size();
  Tensor edge_idx = torch::randint(0, n_edges, { n_pts }, CUDAInt).contiguous();
  Tensor edge_coord = (torch::rand({n_pts, 2}, CUDAFloat) * 2.f - 1.f).contiguous();
  Tensor out_pts = torch::empty({n_pts, 2, 3}, CUDAFloat).contiguous();
  Tensor out_idx = torch::empty({n_pts, 2}, CUDAInt).contiguous();

  dim3 block_dim = LIN_BLOCK_DIM(n_pts);
  dim3 grid_dim  = LIN_GRID_DIM(n_pts);

  GetEdgeSamplesKernel<<<grid_dim, block_dim>>>(n_pts,
                                               RE_INTER(EdgePool*, pers_octree_->edge_pool_gpu_.data_ptr()),
                                               RE_INTER(TransInfo*, pers_octree_->pers_trans_gpu_.data_ptr()),
                                               edge_idx.data_ptr<int>(),
                                               RE_INTER(Wec2f*, edge_coord.data_ptr()),
                                               RE_INTER(Wec3f*, out_pts.data_ptr()),
                                               out_idx.data_ptr<int>());

  return { out_pts, out_idx };
}

__global__ void MarkVistNodeKernel(int n_rays,
                                   int* pts_idx_start_end,
                                   int* oct_indices,
                                   float* sampled_weights,
                                   float* sampled_alpha,
                                   int* visit_weight_adder,
                                   int* visit_alpha_adder,
                                   int* visit_mark,
                                   int* visit_cnt) {
  const int ray_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (ray_idx >= n_rays) { return; }
  const int pts_idx_start = pts_idx_start_end[ray_idx * 2];
  const int pts_idx_end   = pts_idx_start_end[ray_idx * 2 + 1];
  if (pts_idx_start >= pts_idx_end) { return; }
  float max_weight = 0.f;
  float max_alpha = 0.f;
  for (int pts_idx = pts_idx_start; pts_idx < pts_idx_end; pts_idx++) {
    max_weight = fmaxf(max_weight, sampled_weights[pts_idx]);
    max_alpha = fmaxf(max_alpha, sampled_alpha[pts_idx]);
  }

  const float weight_thres = fminf(max_weight * REL_WEIGHT_THRES, ABS_WEIGHT_THRES);
  const float alpha_thres = fminf(max_alpha * REL_ALPHA_THRES, ABS_ALPHA_THRES);

  float cur_oct_weight = 0.f;
  float cur_oct_alpha = 0.f;
  int cur_oct_idx = -1;
  int cur_visit_cnt = 0;
  for (int pts_idx = pts_idx_start; pts_idx < pts_idx_end; pts_idx++) {
    if (cur_oct_idx != oct_indices[pts_idx]) {
      if (cur_oct_idx >= 0) {
        atomicMax(visit_weight_adder + cur_oct_idx, cur_oct_weight > weight_thres ? OCC_WEIGHT_BASE : -1);
        atomicMax(visit_alpha_adder + cur_oct_idx, cur_oct_alpha > alpha_thres ? OCC_ALPHA_BASE : -1);
        atomicMax(visit_cnt + cur_oct_idx, cur_visit_cnt);
        visit_mark[cur_oct_idx] = 1;
      }
      cur_oct_idx = oct_indices[pts_idx];
      cur_oct_weight = 0.f;
      cur_oct_alpha = 0.f;
      cur_visit_cnt = 0;
    }
    cur_oct_weight = fmaxf(cur_oct_weight, sampled_weights[pts_idx]);
    cur_oct_alpha = fmaxf(cur_oct_alpha, sampled_alpha[pts_idx]);
    cur_visit_cnt += 1;
  }
  if (cur_oct_idx >= 0) {
    atomicMax(visit_weight_adder + cur_oct_idx, cur_oct_weight > weight_thres ? OCC_WEIGHT_BASE : -1);
    atomicMax(visit_alpha_adder + cur_oct_idx, cur_oct_alpha > alpha_thres ? OCC_ALPHA_BASE : -1);
    atomicMax(visit_cnt + cur_oct_idx, cur_visit_cnt);
    visit_mark[cur_oct_idx] = 1;
  }
}

__global__ void MarkInvalidNodes(int n_nodes, int* node_weight_stats, int* node_alpha_stats, TreeNode* nodes) {
  int oct_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (oct_idx >= n_nodes) { return; }
  if (node_weight_stats[oct_idx] < 0 || node_alpha_stats[oct_idx] < 0) {
    nodes[oct_idx].trans_idx = -1;
  }
}

void PersSampler::UpdateOctNodes(const SampleResultFlex& sample_result,
                                 const Tensor& sampled_weight,
                                 const Tensor& sampled_alpha) {
  Tensor oct_indices = sample_result.anchors.index({"...", 1}).contiguous();
  const Tensor& sampled_dists = sample_result.dt.contiguous();
  const Tensor& pts_idx_start_end = sample_result.pts_idx_bounds;
  CK_CONT(sampled_dists);
  CK_CONT(oct_indices);
  CK_CONT(sampled_weight);
  CK_CONT(sampled_alpha);
  CK_CONT(pts_idx_start_end);

  const int n_nodes = pers_octree_->tree_nodes_.size();
  const int n_pts = oct_indices.size(0);
  const int n_rays = pts_idx_start_end.size(0);
  CHECK_EQ(n_pts, sampled_weight.size(0));
  CHECK_EQ(n_pts, sampled_alpha.size(0));
  CHECK_EQ(n_pts, sampled_dists.size(0));

  Tensor visit_weight_adder = torch::full({ n_nodes }, -1, CUDAInt);
  Tensor visit_alpha_adder = torch::full({ n_nodes }, -1,CUDAInt);
  Tensor visit_mark = torch::zeros({ n_nodes }, CUDAInt);
  Tensor& visit_cnt = pers_octree_->tree_visit_cnt_;
  CK_CONT(visit_weight_adder);
  CK_CONT(visit_alpha_adder);
  CK_CONT(visit_mark);
  CK_CONT(visit_cnt);

  {
    dim3 block_dim = LIN_BLOCK_DIM(n_rays);
    dim3 grid_dim  = LIN_GRID_DIM(n_rays);
    MarkVistNodeKernel<<<grid_dim, block_dim>>>(n_rays,
                                                pts_idx_start_end.data_ptr<int>(),
                                                oct_indices.data_ptr<int>(),
                                                sampled_weight.data_ptr<float>(),
                                                sampled_alpha.data_ptr<float>(),
                                                visit_weight_adder.data_ptr<int>(),
                                                visit_alpha_adder.data_ptr<int>(),
                                                visit_mark.data_ptr<int>(),
                                                visit_cnt.data_ptr<int>());

  }

  Tensor& node_weight_stats = pers_octree_->tree_weight_stats_;
  Tensor occ_weight_mask = (visit_weight_adder > 0).to(torch::kInt32);
  node_weight_stats = torch::maximum(node_weight_stats, occ_weight_mask * visit_weight_adder);
  node_weight_stats += (visit_mark * (1 - occ_weight_mask) * visit_weight_adder);
  node_weight_stats.clamp_(-100, 1 << 20);
  node_weight_stats = node_weight_stats.contiguous();
  CK_CONT(node_weight_stats);

  Tensor& node_alpha_stats = pers_octree_->tree_alpha_stats_;
  Tensor occ_alpha_mask = (visit_alpha_adder > 0).to(torch::kInt32);
  node_alpha_stats = torch::maximum(node_alpha_stats, occ_alpha_mask * visit_alpha_adder);
  node_alpha_stats += (visit_mark * (1 - occ_alpha_mask) * visit_alpha_adder);
  node_alpha_stats.clamp_(-100, 1 << 20);
  node_alpha_stats = node_alpha_stats.contiguous();
  CK_CONT(node_alpha_stats);

  {
    dim3 block_dim = LIN_BLOCK_DIM(n_nodes);
    dim3 grid_dim  = LIN_GRID_DIM(n_nodes);
    MarkInvalidNodes<<<grid_dim, block_dim>>>(
        n_nodes,
        node_weight_stats.data_ptr<int>(),
            node_alpha_stats.data_ptr<int>(),
        RE_INTER(TreeNode*, pers_octree_->tree_nodes_gpu_.data_ptr()));
  }

  while (!sub_div_milestones_.empty() && sub_div_milestones_.back() <= global_data_pool_->iter_step_) {
    pers_octree_->ProcOctree(true, true, sub_div_milestones_.back() <= 0);
    pers_octree_->MarkInvisibleNodes();
    pers_octree_->ProcOctree(true, false, false);
    sub_div_milestones_.pop_back();
  }

  if (global_data_pool_->iter_step_ % compact_freq_ == 0) {
    pers_octree_->ProcOctree(true, false, false);
  }
}


__device__ int CheckVisible(const Wec3f& center, float side_len,
                            const Watrix33f& intri, const Watrix34f& w2c, const Wec2f& bound) {
  Wec3f cam_pt = w2c * center.homogeneous();
  float radius = side_len * 0.707;
  if (-cam_pt.z() < bound(0) - radius ||
      -cam_pt.z() > bound(1) + radius) {
    return 0;
  }
  if (cam_pt.norm() < radius) {
    return 1;
  }

  float cx = intri(0, 2);
  float cy = intri(1, 2);
  float fx = intri(0, 0);
  float fy = intri(1, 1);
  float bias_x = radius / -cam_pt.z() * fx;
  float bias_y = radius / -cam_pt.z() * fy;
  float img_pt_x = cam_pt.x() / -cam_pt.z() * fx;
  float img_pt_y = cam_pt.y() / -cam_pt.z() * fy;
  if (img_pt_x + bias_x < -cx || img_pt_x > cx + bias_x ||
      img_pt_y + bias_y < -cy || img_pt_y > cy + bias_y) {
    return 0;
  }
  return 1;
}

__global__ void MarkInvisibleNodesKernel(int n_nodes, int n_cams,
                                         TreeNode* tree_nodes,
                                         Watrix33f* intris, Watrix34f* w2cs, Wec2f* bounds) {
  int node_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (node_idx >= n_nodes) { return; }
  int n_visible_cams = 0;
  for (int cam_idx = 0; cam_idx < n_cams; cam_idx++) {
    n_visible_cams += CheckVisible(tree_nodes[node_idx].center,
                                   tree_nodes[node_idx].side_len,
                                   intris[cam_idx],
                                   w2cs[cam_idx],
                                   bounds[cam_idx]);
  }
  if (n_visible_cams < 1) {
    tree_nodes[node_idx].trans_idx = -1;
  }
}

void PersOctree::MarkInvisibleNodes() {
  int n_nodes = tree_nodes_.size();
  int n_cams = intri_.size(0);

  CK_CONT(intri_);
  CK_CONT(w2c_);
  CK_CONT(bound_);

  dim3 block_dim = LIN_BLOCK_DIM(n_nodes);
  dim3 grid_dim = LIN_GRID_DIM(n_nodes);
  MarkInvisibleNodesKernel<<<grid_dim, block_dim>>>(
      n_nodes, n_cams,
      RE_INTER(TreeNode*, tree_nodes_gpu_.data_ptr()),
      RE_INTER(Watrix33f*, intri_.data_ptr()),
      RE_INTER(Watrix34f*, w2c_.data_ptr()),
      RE_INTER(Wec2f*, bound_.data_ptr())
  );
}